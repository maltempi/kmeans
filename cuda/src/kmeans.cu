#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "point.h"
#include "config.h"

/**
    Groups the points in a centroid.
*/
__global__ void km_group_by_cluster(Point* points, Centroid* centroids,
        int num_centroids, int num_points)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int i = 0;

    float minor_distance = -1.0;

	if (idx < num_points) {
	    for (i = 0; i < num_centroids; i++) {
	        float my_distance = km_distance(&points[idx], &centroids[i]);
	
	        // if my_distance is less than the lower minor_distance 
	        // or minor_distance is not yet started
	        if (minor_distance > my_distance || minor_distance == -1.0) {
	            minor_distance = my_distance;
	            points[idx].cluster = i;
	        }
	    }
	}
}

/**
    Sum the points of each centroid
*/
__global__ void km_sum_points_cluster(Point* points, Centroid* centroids,
        int num_centroids, int num_points)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_points) {
	    for (int i = 0; i < num_centroids; i++) {
	        if (points[idx].cluster == i) {
	            atomicAdd(&centroids[i].x_sum, points[idx].x);
	            atomicAdd(&centroids[i].y_sum, points[idx].y);
	            atomicAdd(&centroids[i].num_points, 1);
	        }
	    }
	}
}

/**
    Clear the x_sum, y_sum and num_points, used in last iteration.
*/
__global__ void km_clear_last_iteration(Centroid* centroids, int num_centroids)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx < num_centroids) {
	    centroids[idx].x_sum = 0.0;
	    centroids[idx].y_sum = 0.0;
	    centroids[idx].num_points = 0.0;
    }
}

/**
    Update the centroids with current clustering.
    Gets the x and y sum and divides by number of point for each centroid.\
*/
__global__ void km_update_centroids(Centroid* centroids, int num_centroids)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < num_centroids) {
	    if (centroids[idx].num_points > 0) {
	        centroids[idx].x = centroids[idx].x_sum / centroids[idx].num_points;
	        centroids[idx].y = centroids[idx].y_sum / centroids[idx].num_points;
	    }
	}
    // I need this values to plot, so, I created km_clear_last_iteration.
    // with this new function we lost 1ms :'(
    // __syncthreads();
    // clear the values to next iteration
    // centroids[idx].x_sum = 0.0;
    // centroids[idx].y_sum = 0.0;
    // centroids[idx].num_points = 0.0;
}

/**
    Compare the clusters of each point.
    @param p1 - points of current iteration
    @param p2 - points of last iteration
*/
__global__ void km_points_compare(Point* p1, Point* p2, int num_points,
        int *result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        // if any points has its cluster different, changes the result variable
        if (p1[idx].cluster != p2[idx].cluster) {
            *result = 0;
        }
    }
}

/**
    Copy a point array.
    Utilized to copy the status of points on the last iteration to compare them.
*/
__global__ void km_points_copy(Point* p_dest, Point* p_src, int num_points)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        p_dest[idx] = p_src[idx];
    }
}

/**
* Executes the k-mean algorithm.
* To measure your global methods, use that:
*
*    hipEvent_t start, stop;
*    float time;
*    hipEventCreate(&start);
*    hipEventCreate(&stop);
*    hipEventRecord(start, 0);
*
* //  put your__global__ method here!
*
*    hipEventRecord(stop, 0);
*    hipEventSynchronize(stop);
*    hipEventElapsedTime(&time, start, stop);
*    printf("%lf\n", times)
*/
void km_execute(Point* h_points, Centroid* h_centroids, int num_points,
        int num_centroids)
{
    int iterations = 0;
    Point* d_points;
    Point* d_points_old;
    Centroid* d_centroids;
    int h_res = 1;
    int *d_res;

    hipMalloc((void**) &d_res, sizeof(int));
    hipMalloc((void**) &d_points_old, sizeof(Point) * num_points);
    hipMalloc((void **) &d_points, sizeof(Point) * num_points);
    hipMalloc((void **) &d_centroids, sizeof(Centroid) * num_centroids);

    hipMemcpy(d_points, h_points, sizeof(Point) * num_points, hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, h_centroids, sizeof(Centroid) * num_centroids, hipMemcpyHostToDevice);   

    while (true) {

        km_clear_last_iteration<<<ceil(num_centroids/10), 10>>>(d_centroids, num_centroids);
        hipDeviceSynchronize();

        km_group_by_cluster<<<ceil(num_points/100), 100>>>(d_points, d_centroids,
                num_centroids, num_points);
        hipDeviceSynchronize();
        
        km_sum_points_cluster<<<ceil(num_points/100), 100>>>(d_points, d_centroids,
                num_centroids, num_points);
        hipDeviceSynchronize();

        km_update_centroids<<<ceil(num_centroids/10), 10>>>(d_centroids, num_centroids);
        hipDeviceSynchronize();

        if (REPOSITORY_SPECIFICATION == 1) {
            // in repository specifications, 
            // we just want know if number of 
            // iterations is equals NUMBER_OF_ITERATIONS - 1 (iterations starts in 0)
            if (iterations == (NUMBER_OF_ITERATIONS - 1)) {
                break;
            }
        } else {
            // TODO: WARNING:
            // THIS IMPLEMENTATION IS NOT WORKING YET!
            if (iterations > 0) {
                h_res = 1;
                hipMemcpy(d_res, &h_res , sizeof(int), hipMemcpyHostToDevice);
                km_points_compare<<<ceil(num_points/10), 10>>>(d_points, d_points_old,
                        num_points, d_res);
                hipDeviceSynchronize();

                hipMemcpy(&h_res, d_res, sizeof(int), hipMemcpyDeviceToHost);

                // if h_rest == 1 the two vector of points are equal and the kmeans iterations
                // has completed all work
                if (h_res == 1) {
                    break;
                }
            }

            km_points_copy<<<ceil(num_points/100), 100>>>(d_points_old, d_points,
                num_points);
            hipDeviceSynchronize();
        }
        
        iterations++;
    }

    hipMemcpy(h_centroids, d_centroids , sizeof(Centroid) * num_centroids, hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_points_old);
    hipFree(d_res);
}
